#include "hip/hip_runtime.h"
#include "cuda/symmetric_functions.hpp"

namespace cuda{

    __global__ void calculate_sf_kernel(
                    const at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> cartesians,
                    at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> g_total,
                    const int* features, float** params, int n_atoms, int n_features
    );

    __device__ float G1(const float rij, const float rc);
    __device__ float G2(const float rij, const float rc, const float eta, const float rs);
    __device__ float G3(const float rij, const float rc, const float k);
    __device__ float G4(const float rij, const float rik, const float rjk, const float rc,
            const float eta, const float lambda, const float xi,
            const float cos_v);
    __device__ float G5(const float rij, const float rik, const float rjk, const float rc,
            const float eta, const float lambda, const float xi,
            const float cos_v);

    // @brief Calculates symmetry functions
    // @param cartesians: atomic positions
    // @param features: list of symmetry functions to calculate
    // @param params: list of parameters for each symmetry function
    // (length of list must be equal to length of features list
    // and all vectors must have the size equal to the number of symmetry functions params)
    // @return g_total: symmetry functions
    Tensor calculate_sf(const Tensor& cartesians, const vector<int>& features, const vector<vector<float>>& params){
        CHECK_INPUT(cartesians);
        
        torch::TensorOptions opts = torch::TensorOptions()
                                        .dtype(torch::kFloat)
                                        .device(torch::kCUDA);
        int N_atoms = cartesians.size(0);
        int N_features = features.size();
        
        // output g values
        Tensor g_total = torch::zeros({N_atoms, N_features}, opts);

        int threads = 512;
        dim3 blocks(N_atoms, N_features, N_atoms);

        // accessors to torch tensors for gpu
        auto cartesians_accessor = cartesians.packed_accessor32<float, 2, torch::RestrictPtrTraits>();
        auto g_total_accessor = g_total.packed_accessor32<float, 2, torch::RestrictPtrTraits>();

        // copy features and params to device
        int* features_d;
        hipMalloc(&features_d, N_features * sizeof(int));
        hipMemcpy(features_d, features.data(), N_features * sizeof(int), hipMemcpyHostToDevice);

        float** params_d;
        float** params_h = (float**)malloc(N_features * sizeof(float*));
        for (int i = 0; i < N_features; i++){
            hipMalloc(&params_h[i], params[i].size() * sizeof(float));
            hipMemcpy(params_h[i], params[i].data(), params[i].size() * sizeof(float), hipMemcpyHostToDevice);
        }
        hipMalloc(&params_d, N_features * sizeof(float*));
        hipMemcpy(params_d, params_h, N_features * sizeof(float*), hipMemcpyHostToDevice);
        free(params_h);

        calculate_sf_kernel<<<blocks, threads>>>(
                cartesians_accessor, g_total_accessor,
                features_d, params_d, N_atoms, N_features
        );

        // normalize g values
        g_total = torch::nn::functional::normalize(g_total,
                                                torch::nn::functional::NormalizeFuncOptions()
                                                                        .p(2.0)
                                                                        .dim(1));
        return g_total;
    }

    __global__ void calculate_sf_kernel(
                    const at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> cartesians,
                    at::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> g_total,
                    const int* features, float** params, int n_atoms, int n_features
    ){

            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int feature_index = blockIdx.y;
            int j = blockIdx.z * blockDim.z + threadIdx.z;

            float g;
            if(i < n_atoms && feature_index < n_features){
                    switch (features[feature_index]){
                        // G1
                        case 1:{
                            if(j < n_atoms){
                                if (i == j){
                                    break;
                                }
                                auto ri = cartesians[i];
                                auto rj = cartesians[j];
                                float rij = 0;
                                for (int dim = 0; dim < 3; dim++){
                                    rij += (ri[dim] - rj[dim]) * (ri[dim] - rj[dim]);
                                }

                                rij = sqrt(rij);
                                g = G1(rij, params[feature_index][0]);
                            }
                            break;
                        }
                        // G2
                        case 2:
                        {
                            if(j < n_atoms){
                                if (i == j){
                                    break;
                                }
                                auto ri = cartesians[i];
                                auto rj = cartesians[j];
                                float rij = 0;
                                for (int dim = 0; dim < 3; dim++){
                                    rij += (ri[dim] - rj[dim]) * (ri[dim] - rj[dim]);
                                }
                                rij = sqrt(rij);
                                g = G2(rij, params[feature_index][0], params[feature_index][1], params[feature_index][2]);
                            }
                            break;
                        }
                        // G3
                        case 3:
                        {
                            if(j < n_atoms){
                                if (i == j){
                                    break;
                                }
                                auto ri = cartesians[i];
                                auto rj = cartesians[j];
                                float rij = 0;
                                for (int dim = 0; dim < 3; dim++){
                                    rij += (ri[dim] - rj[dim]) * (ri[dim] - rj[dim]);
                                }
                                rij = sqrt(rij);
                                g = G3(rij, params[feature_index][0], params[feature_index][1]);
                            }
                            break;
                        }
                        // G4
                        case 4:
                        {
                            if (j < n_atoms){
                                for (int k = 0; k < n_atoms; k++){
                                    if (i == j || i == k || j == k){
                                        continue;
                                    }
                                    auto ri = cartesians[i];
                                    auto rj = cartesians[j];
                                    auto rk = cartesians[k];

                                    float rij = 0;
                                    float rik = 0;
                                    float rjk = 0;

                                    for (int dim = 0; dim < 3; dim++){
                                        rij += (ri[dim] - rj[dim]) * (ri[dim] - rj[dim]);
                                        rjk += (rk[dim] - rj[dim]) * (rk[dim] - rj[dim]);
                                        rik += (rk[dim] - ri[dim]) * (rk[dim] - ri[dim]);
                                    }

                                    rij = sqrt(rij);
                                    rik = sqrt(rik);
                                    rjk = sqrt(rjk);

                                    float cos_v = (rij * rij + rik * rik - rjk * rjk) / 2 / rij / rik;

                                    g = G4(rij, rik, rjk,
                                             params[feature_index][0], params[feature_index][1], 
                                             params[feature_index][2], params[feature_index][3], cos_v);
                                }
                            }
                            break;
                        }
                        // G5
                        case 5:
                        {
                            if (j < n_atoms){
                                for (int k = 0; k < n_atoms; k++){
                                    if (i == j || i == k || j == k){
                                        continue;
                                    }

                                    auto ri = cartesians[i];
                                    auto rj = cartesians[j];
                                    auto rk = cartesians[k];

                                    float rij = 0;
                                    float rik = 0;
                                    float rjk = 0;
                                    for (int dim = 0; dim < 3; dim++){
                                        rij += (ri[dim] - rj[dim]) * (ri[dim] - rj[dim]);
                                        rjk += (rk[dim] - rj[dim]) * (rk[dim] - rj[dim]);
                                        rik += (rk[dim] - ri[dim]) * (rk[dim] - ri[dim]);
                                    }
                                    rij = sqrt(rij);
                                    rik = sqrt(rik);
                                    rjk = sqrt(rjk);
                                    float cos_v = (rij * rij + rik * rik - rjk * rjk) / 2 / rij / rik;
                                    
                                    g = G5(rij, rik, rjk,
                                             params[feature_index][0], params[feature_index][1], 
                                             params[feature_index][2], params[feature_index][3], cos_v);
                                }
                            }
                            break;
                        }
                    }
                    g_total[i][feature_index - 1] += g;
                }
    }

    __device__ float cutf(const float rij, const float rc){
        if (rij < rc){
            return 0.5 * (cos(M_PI * rij / rc) + 1);
        }
        return 0;
    }

    __device__ float dcutf(const float rij, const float rc){
        if (rij < rc){
            return 0.5 * (-M_PI * sin(M_PI * rij / rc) / rc);
        }
        return 0;
    }

    __device__ float G1(const float rij, const float rc){
        return cutf(rij, rc);
    }

    __device__ float G2(const float rij, const float rc, const float eta, const float rs){
        return exp(-eta * (rij - rs) * (rij - rs)) * cutf(rij, rc);
    }

    __device__ float G3(const float rij, const float rc, const float kappa){ 
        return cos(kappa * rij) * cutf(rij, rc);
    }

    __device__ float G4(const float rij, const float rik, const float rjk, const float rc,
            const float eta, const float lambda, const float zeta,
            const float cos_v){ 
            
        float out_g;
        float expv = exp(-eta * (rij * rij + rik * rik + rjk * rjk)); 
        float cosv = 1 + lambda * cos_v;
        float powcos;
        if (fabs(cosv) < 10e-4){
            powcos = 0;
        }
        else{
            powcos = pow(cosv, zeta);
        }
        out_g = pow(2, 1 - zeta) * powcos * expv * \
                cutf(rij, rc) * cutf(rik, rc) * cutf(rjk, rc);
                
        return out_g;
    }

    __device__ float G5(const float rij, const float rik, const float rjk, const float rc,
            const float eta, const float lambda, const float zeta,
            const float cos_v){ 
            
        float out_g;
        float expv = exp(-eta * (rij * rij + rik * rik)); 
        float cosv = 1 + lambda * cos_v;
        float powcos;
        if (fabs(cosv) < 10e-4){
            powcos = 0;
        }
        else{
            powcos = pow(cosv, zeta);
        }
        out_g = pow(2, 1 - zeta) * powcos * expv * \
                cutf(rij, rc) * cutf(rik, rc);

        return out_g;
    }
}